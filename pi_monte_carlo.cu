
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

__global__ 
void count_samples_in_circles(float* d_randNumsX, float* d_randNumsY, int* d_countInBlocks, int num_blocks, int nsamples) 
{

  __shared__ int shared_blocks[500];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * num_blocks;

  int inCircle = 0;  
  for (int i = index; i < nsamples; i+= stride) {    
    float xValue = d_randNumsX[i];    
    float yValue = d_randNumsY[i];

    if (xValue*xValue + yValue*yValue <= 1.0f) {
      inCircle++;    
    }  
  }
  
  shared_blocks[threadIdx.x] = inCircle;

  __syncthreads();

  if (threadIdx.x == 0) 
  {    
    int totalInCircleForABlock = 0;    
    for (int j = 0; j < blockDim.x; j++) 
    {      
      totalInCircleForABlock += shared_blocks[j];    
    }
    d_countInBlocks[blockIdx.x] = totalInCircleForABlock;  
  }
}

int nsamples;

int main(int argc, char* argv[]) {   
    int nsamples = atoi(argv[1]);
    printf("nsamples: %d\n", nsamples);   
  vector<float> h_randNumsX(nsamples);    
  vector<float> h_randNumsY(nsamples);
  srand(time(NULL));    
  for (int i = 0; i < h_randNumsX.size(); ++i) 
  {        
    h_randNumsX[i] = float(rand()) / RAND_MAX;        
    h_randNumsY[i] = float(rand()) / RAND_MAX;    
  }
    
  size_t size = nsamples * sizeof(float);    
  float* d_randNumsX;    
  float* d_randNumsY;    
  
  hipMalloc(&d_randNumsX, size);  
  hipMalloc(&d_randNumsY, size);
  hipMemcpy(d_randNumsX, &h_randNumsX.front(), size, hipMemcpyHostToDevice);    
  hipMemcpy(d_randNumsY, &h_randNumsY.front(), size, hipMemcpyHostToDevice);
  
  int threadsPerBlock = 500;
  int num_blocks = nsamples / (1000 * threadsPerBlock);
  size_t countBlocks = num_blocks * sizeof(int);
 
  int* d_countInBlocks;
  hipMalloc(&d_countInBlocks, countBlocks);

  count_samples_in_circles<<<num_blocks, threadsPerBlock>>>(d_randNumsX, d_randNumsY, d_countInBlocks, num_blocks, nsamples);
  if ( hipSuccess != hipGetLastError() )
    cout << "Error!\n";

  int* h_countInBlocks = new int[num_blocks];
  hipMemcpy(h_countInBlocks, d_countInBlocks, countBlocks, hipMemcpyDeviceToHost);

  int nsamples_in_circle = 0;
  for (int i = 0 ; i < num_blocks; i++) {
    nsamples_in_circle = nsamples_in_circle + h_countInBlocks[i];
  }

  hipFree(d_randNumsX);
  hipFree(d_randNumsY);
  hipFree(d_countInBlocks);

  float estimatedValue = 4.0 * float(nsamples_in_circle) / nsamples;

  cout << "Estimated Value: " << estimatedValue << endl;
}
